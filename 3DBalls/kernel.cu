
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <GL\glut.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

#define N  500

GLdouble eyeX = 0, eyeY = 0, eyeZ = 5;
GLdouble centerX = 0, centerY = 0, centerZ = 0;
GLdouble upX = 0, upY = 1, upZ = 0;

GLfloat xRotated = 0, yRotated = 0, zRotated = 0;
GLdouble radius = 0.01;

GLfloat ballsCoordinates[N * 3];

/*************FUNCTIONS*****************/
/***************************************/
void display(void);
void reshape(int x, int y);
void initData();
float getRandomCord();

void drawBackFace();
void drawFrontFace();
void drawLeftFace();
void drawRightFace();
void drawBottomFace();

void specialKeys(int key, int x, int y);

/************   MAIN   *****************/
/***************************************/
int main(int argc, char **argv)
{
	initData();

	glutInit(&argc, argv);
	glutInitWindowSize(1000, 1000);
	glutCreateWindow("Solid Sphere");
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(display);
	glutSpecialFunc(specialKeys);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}


/******  Functions Declaration  ********/
/***************************************/
void display(void)
{

	glMatrixMode(GL_MODELVIEW);
	
	// clear the drawing buffer.
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	// clear the identity matrix.
	glLoadIdentity();
	glEnable(GL_BLEND);
	gluLookAt(eyeX, eyeY, eyeZ,
		centerX, centerY, centerZ,
		upX, upY, upZ);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	
	glRotatef(xRotated, 1.0, 0.0, 0.0);
	// rotation about Y axis
	glRotatef(yRotated, 0.0, 1.0, 0.0);
	// rotation about Z axis
	//glRotatef(zRotated, 0.0, 0.0, 1.0);
	

	for (int i = 0; i < N; i++){
		glPushMatrix();
		// traslate the draw by z = -4.0
		// Note this when you decrease z like -8.0 the drawing will looks far , or smaller.
		glTranslatef(ballsCoordinates[i], ballsCoordinates[i+1], ballsCoordinates[i+2]);
		// Red color used to draw.
		glColor3f(0.9, 0.3, 0.2);
		// changing in transformation matrix.
		
		glScalef(1.0, 1.0, 1.0);
		// built-in (glut library) function , draw you a sphere.
		glutWireSphere(radius, 20, 20);
		glPopMatrix();
		// Flush buffers to screen
	}
	int currentAngle = (int)yRotated % 360;
	if (currentAngle < 0){
		currentAngle = 360 + currentAngle;
	}
	if ((45 >= currentAngle && currentAngle>=0) || (360 >= currentAngle && currentAngle>315)){
		drawBackFace();
		drawLeftFace();
		drawRightFace();
		drawFrontFace();
	}
	else if (45 < currentAngle && currentAngle <= 135){
		drawRightFace();
		drawFrontFace();
		drawBackFace();
		drawLeftFace();
			
	}
	else if (135 < currentAngle && currentAngle <= 225){
		drawFrontFace();
		drawRightFace();
		drawLeftFace();
		drawBackFace();
	}
	else{
		drawLeftFace();
		drawFrontFace();
		drawBackFace();
		drawRightFace();
	}
	glFlush();
	// sawp buffers called because we are using double buffering 
	// glutSwapBuffers();
}

void drawBackFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, -1.0);       // P1
	glVertex3f(-1.0, 1.0, -1.0);       // P2
	glVertex3f(1.0, 1.0, -1.0);       // P3
	glVertex3f(1.0, -1.0, -1.0);       // P4
	glEnd();
}
void drawFrontFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, 1.0);       // P1
	glVertex3f(-1.0, 1.0, 1.0);       // P2
	glVertex3f(1.0, 1.0, 1.0);       // P3
	glVertex3f(1.0, -1.0, 1.0);       // P4
	glEnd();
}
void drawLeftFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, 1.0);       // P1
	glVertex3f(-1.0, 1.0, 1.0);       // P2
	glVertex3f(-1.0, 1.0, -1.0);       // P3
	glVertex3f(-1.0, -1.0, -1.0);       // P4
	glEnd();
}
void drawRightFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(1.0, -1.0, 1.0);       // P1
	glVertex3f(1.0, 1.0, 1.0);       // P2
	glVertex3f(1.0, 1.0, -1.0);       // P3
	glVertex3f(1.0, -1.0, -1.0);       // P4
	glEnd();
}
void reshape(int x, int y)
{
	if (y == 0 || x == 0) return;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(39.0, (GLdouble)x / (GLdouble)y, 0.6, 21.0);
	glMatrixMode(GL_MODELVIEW);
	glViewport(0, 0, x, y);  //Use the whole window for rendering
} 

void initData(){
	for (int i = 0; i < N; i++)
	{
		ballsCoordinates[i] = getRandomCord();
		ballsCoordinates[i + 1] = getRandomCord();
		ballsCoordinates[i + 2] = getRandomCord();
	}
}

float getRandomCord()
{
	int c = rand() % 4;
	float r = -1.0f + (rand() / (float)RAND_MAX * 2.0f);
	r = r + (c * 0.000005f);
	return r;
}

void specialKeys(int key, int x, int y) {

	//  Right arrow - increase rotation by 5 degree
	if (key == GLUT_KEY_RIGHT){
		yRotated += 1;
	}
		
	//  Left arrow - decrease rotation by 5 degree
	else if (key == GLUT_KEY_LEFT){
		yRotated -= 1;
	}
	else if (key == GLUT_KEY_UP){
		xRotated += 1;
	}

	else if (key == GLUT_KEY_DOWN){
		xRotated -= 1;
	}
	else if (key == GLUT_KEY_PAGE_UP){
		eyeZ += 0.05;
	}
	else if (key == GLUT_KEY_PAGE_DOWN){
		eyeZ -= 0.05;
	}

	//  Request display update
	glutPostRedisplay();

}

