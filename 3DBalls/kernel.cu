#include "hip/hip_runtime.h"
#include <windows.h> 
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <GL\glut.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>



#define N  10
#define GRAVITY 0.01
#define SPRINGINESS 0.95
#define RADIUS 0.08

bool isCalculatedOnGPU = true;
float colorStep = 1.0 / N;

int drawCallsCount = 0;
int measureCount = 1;
float timeSum = 0;
int refreshMillis = 35;


GLdouble eyeX = 0, eyeY = 0, eyeZ = 5;
GLdouble centerX = 0, centerY = 0, centerZ = 0;
GLdouble upX = 0, upY = 1, upZ = 0;

GLfloat xRotated = 45, yRotated = 45, zRotated = 45;

GLfloat ballsCoordinates[N * 3];
/*************    HOST   **************/

GLfloat speedTable_H[N * 3];
bool collisionMatrix_H[N * N];
int collisionSafetyCounter_H[N * N];

/*************   DEVICE   **************/

__device__ GLfloat speedTable_C[N * 3];
__device__ bool collisionMatrix[N * N];
__device__ int collisionSafetyCounter[N * N];

/*************FUNCTIONS*****************/
/***************************************/
void display(void);
void reshape(int x, int y);
void Timer(int value);

void initData();
float getRandomCord();
float getRandomSpeed();
double second();

void drawBackFace();
void drawFrontFace();
void drawLeftFace();
void drawRightFace();
void drawBottomFace();

void calculateNewPositionsCPU(float* ballsTable);
int detectCollisionCPU(GLfloat x, GLfloat y, GLfloat z, int ballNumber, GLfloat * ballTable);

void specialKeys(int key, int x, int y);

__global__ void initGpuData(float* speedTable);
hipError_t sendDataToGPU();
__global__ void calculateNewPositions(float* ballsTable, float springiness, float radius);
hipError_t sendAndCalculateCordsOnGPU(float* ballTable);
__device__ int detectCollision(GLfloat x, GLfloat y, GLfloat z, int ballNumber, GLfloat * ballTable);

/************   MAIN   *****************/
/***************************************/
int main(int argc, char **argv)
{
	initData();
	sendDataToGPU();

	glutInit(&argc, argv);
	glutInitWindowSize(1000, 1000);
	glutCreateWindow("3DBalls");
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(display);
	glutSpecialFunc(specialKeys);
	glutReshapeFunc(reshape);
	glutTimerFunc(0, Timer, 0);
	glutMainLoop();
	return 0;
}


/******  Functions Declaration  ********/
/***************************************/
void display(void)
{

	glMatrixMode(GL_MODELVIEW);
	
	// clear the drawing buffer.
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	// clear the identity matrix.
	glLoadIdentity();
	glEnable(GL_BLEND);
	gluLookAt(eyeX, eyeY, eyeZ,
		centerX, centerY, centerZ,
		upX, upY, upZ);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	
	glRotatef(xRotated, 1.0, 0.0, 0.0);
	// rotation about Y axis
	glRotatef(yRotated, 0.0, 1.0, 0.0);
	// rotation about Z axis
	//glRotatef(zRotated, 0.0, 0.0, 1.0);
	

	for (int i = 0; i < N; i++){
		glPushMatrix();
		// traslate the draw by z = -4.0
		// Note this when you decrease z like -8.0 the drawing will looks far , or smaller.
		glTranslatef(ballsCoordinates[i*3], ballsCoordinates[i*3+1], ballsCoordinates[i*3+2]);
		// Red color used to draw.
		glColor3f(0.9, colorStep*i, 0.2);
		// changing in transformation matrix.
		
		glScalef(1.0, 1.0, 1.0);
		// built-in (glut library) function , draw you a sphere.
		glutWireSphere(RADIUS, 20, 20);
		glPopMatrix();
		// Flush buffers to screen
	}
	int currentAngle = (int)yRotated % 360;
	if (currentAngle < 0){
		currentAngle = 360 + currentAngle;
	}
	if ((45 >= currentAngle && currentAngle>=0) || (360 >= currentAngle && currentAngle>315)){
		drawBackFace();
		drawLeftFace();
		drawRightFace();
		drawFrontFace();
	}
	else if (45 < currentAngle && currentAngle <= 135){
		drawRightFace();
		drawFrontFace();
		drawBackFace();
		drawLeftFace();
			
	}
	else if (135 < currentAngle && currentAngle <= 225){
		drawFrontFace();
		drawRightFace();
		drawLeftFace();
		drawBackFace();
	}
	else{
		drawLeftFace();
		drawFrontFace();
		drawBackFace();
		drawRightFace();
	}
	glFlush();
	// sawp buffers called because we are using double buffering 
	// glutSwapBuffers();

	if (isCalculatedOnGPU){
		drawCallsCount++;
		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		hipError_t cudaStatus = sendAndCalculateCordsOnGPU(ballsCoordinates);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&time, start, stop);
		timeSum += time;
		if (drawCallsCount == 100)
		{
			printf("Average fps %f . Draw calls count: %d \n", (float)(1000 / (timeSum / 100)), (measureCount * 100));
			drawCallsCount = 0;
			measureCount++;
			timeSum = 0;
		}
	}
	else{
		drawCallsCount++;
		float hostTime;
		double startTime, stopTime, elapsed;
		startTime = second();
		calculateNewPositionsCPU(ballsCoordinates);
		stopTime = second();
		hostTime = (stopTime - startTime) * 1000;
		timeSum += hostTime;
		if (drawCallsCount == 100)
		{
			printf("Average fps %f . Draw calls count: %d \n", (float)(1000 / (timeSum / 100)), (measureCount * 100));
			drawCallsCount = 0;
			measureCount++;
			timeSum = 0;
		}
	}
}

void drawBackFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, -1.0);       // P1
	glVertex3f(-1.0, 1.0, -1.0);       // P2
	glVertex3f(1.0, 1.0, -1.0);       // P3
	glVertex3f(1.0, -1.0, -1.0);       // P4
	glEnd();
}
void drawFrontFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, 1.0);       // P1
	glVertex3f(-1.0, 1.0, 1.0);       // P2
	glVertex3f(1.0, 1.0, 1.0);       // P3
	glVertex3f(1.0, -1.0, 1.0);       // P4
	glEnd();
}
void drawLeftFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(-1.0, -1.0, 1.0);       // P1
	glVertex3f(-1.0, 1.0, 1.0);       // P2
	glVertex3f(-1.0, 1.0, -1.0);       // P3
	glVertex3f(-1.0, -1.0, -1.0);       // P4
	glEnd();
}
void drawRightFace(){
	glBegin(GL_POLYGON);
	glColor4f(0, 0, 1, 0.5);
	glVertex3f(1.0, -1.0, 1.0);       // P1
	glVertex3f(1.0, 1.0, 1.0);       // P2
	glVertex3f(1.0, 1.0, -1.0);       // P3
	glVertex3f(1.0, -1.0, -1.0);       // P4
	glEnd();
}
void reshape(int x, int y)
{
	if (y == 0 || x == 0) return;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(39.0, (GLdouble)x / (GLdouble)y, 0.6, 21.0);
	glMatrixMode(GL_MODELVIEW);
	glViewport(0, 0, x, y);  //Use the whole window for rendering
} 

void Timer(int value) {
	glutPostRedisplay();	// Post a paint request to activate display()
	glutTimerFunc(refreshMillis, Timer, 0); // subsequent timer call at milliseconds
}

void initData(){
	for (int i = 0; i < N; i++)
	{
		ballsCoordinates[i*3] = getRandomCord();
		ballsCoordinates[i*3 + 1] = getRandomCord();
		ballsCoordinates[i*3 + 2] = getRandomCord();
	}
	/*for (int i = 0; i < N * 3; i++){
		speedTable_C[i] = speedTable_H[i];
	}*/
	for (int i = 0; i < N; i++){
		speedTable_H[i * 3] = getRandomSpeed();
		speedTable_H[i * 3 + 1] = 0;
		speedTable_H[i * 3 + 2] = getRandomSpeed();
	}
}

float getRandomCord()
{
	int c = rand() % 4;
	float r = -1.0f + (rand() / (float)RAND_MAX * 2.0f);
	r = r + (c * 0.000005f);
	return r;
}

float getRandomSpeed()
{
	int c = rand() % 2;
	float a = 0.1f;
	float r = ((rand() / (float)RAND_MAX * a));
	if (c == 1)
		r = -r;
	return r;
}
double second()
{
	LARGE_INTEGER t;
	static double oofreq;
	static int checkedForHighResTimer;
	static BOOL hasHighResTimer;
	if (!checkedForHighResTimer) {
		hasHighResTimer = QueryPerformanceFrequency(&t);
		oofreq = 1.0 / (double)t.QuadPart;
		checkedForHighResTimer = 1;
	}
	if (hasHighResTimer) {
		QueryPerformanceCounter(&t);
		return (double)t.QuadPart * oofreq;
	}
	else {
		return (double)GetTickCount() / 1000.0;
	}
}

__global__ void initGpuData(float* speedTable){
	for (int i = 0; i < N * 3; i++){
		speedTable_C[i] = speedTable[i];
	}
	for (int i = 0; i < N*N; i++){
		collisionMatrix[i] = false;
		collisionSafetyCounter[i] = 0;
	}
}
hipError_t sendDataToGPU(){
	float speedTable[3 * N];
	float* dev_speedTable = 0;
	hipError_t cudaStatus;

	for (int i = 0; i < N; i++){
		speedTable[i * 3] = getRandomSpeed();
		speedTable[i * 3 + 1] = 0;
		speedTable[i * 3 + 2] = getRandomSpeed();
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_speedTable, 3 * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_speedTable, speedTable,3 * N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	initGpuData << <1, 1 >> >(dev_speedTable);

Error:
	hipFree(dev_speedTable);

	return cudaStatus;
}

__global__ void calculateNewPositions(float* ballsTable){
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	while (k < N){
		if (k < N) {			
			ballsTable[k * 3] += speedTable_C[k * 3];
			ballsTable[k * 3 + 1] += speedTable_C[k * 3 + 1];
			ballsTable[k * 3 + 2] += speedTable_C[k * 3 + 2];
			// Check if the ball exceeds the edges
			if (ballsTable[k * 3] > 1.0 - RADIUS){
				ballsTable[k * 3] = 1.0 - RADIUS;
				speedTable_C[k * 3] = -speedTable_C[k * 3] * SPRINGINESS;		
			}
			if (ballsTable[k * 3] < -1.0 + RADIUS){
				ballsTable[k * 3] = -1.0 + RADIUS;
				speedTable_C[k * 3] = -speedTable_C[k * 3] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 1] > 1.0 - RADIUS){
				ballsTable[k * 3 + 1] = 1.0 - RADIUS;
				speedTable_C[k * 3 + 1] = -speedTable_C[k * 3 + 1] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 1] < -1.0 + RADIUS){
				ballsTable[k * 3 + 1] = -1.0 + RADIUS;
				speedTable_C[k * 3 + 1] = -speedTable_C[k * 3 + 1] * SPRINGINESS;
			}			

			if (ballsTable[k * 3 + 2] > 1.0 - RADIUS){
				ballsTable[k * 3 + 2] = 1.0 - RADIUS;
				speedTable_C[k * 3 + 2] = -speedTable_C[k * 3 + 2] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 2] < -1.0 + RADIUS){
				ballsTable[k * 3 + 2] = -1.0 + RADIUS;
				speedTable_C[k * 3 + 2] = -speedTable_C[k * 3 + 2] * SPRINGINESS;
			}

			int ballDetected = detectCollision(ballsTable[k * 3], ballsTable[k * 3 + 1], ballsTable[k * 3 + 2], k, ballsTable);

			if (ballDetected != -1){
				float tmpSpeedX = speedTable_C[k * 3];
				float tmpSpeedY = speedTable_C[k * 3 + 1];
				float tmpSpeedZ = speedTable_C[k * 3 + 2];
				speedTable_C[k * 3] = speedTable_C[ballDetected * 3];
				speedTable_C[k * 3 + 1] = speedTable_C[ballDetected * 3 + 1];
				speedTable_C[k * 3 + 2] = speedTable_C[ballDetected * 3 + 2];
				speedTable_C[ballDetected * 3] = tmpSpeedX;
				speedTable_C[ballDetected * 3 + 1] = tmpSpeedY;
				speedTable_C[ballDetected * 3 + 2] = tmpSpeedZ;
			}
			//FRICTION
			if ((ballsTable[k * 3 + 1] < -1.0 + RADIUS + 0.0003) && (speedTable_C[k * 3 + 1] < 0.02)){
				speedTable_C[k * 3] *= 0.98;
				speedTable_C[k * 3 + 2] *= 0.98;
			}
			//gravity
			
			speedTable_C[k*3+1] -= GRAVITY;
			//tmpSpeedTableY[k] -= 0.01f;
		}
		k += blockDim.x * gridDim.x;
	}
}
hipError_t sendAndCalculateCordsOnGPU(float* ballTable)
{
	float* dev_ballTable = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_ballTable, 3 * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_ballTable , ballTable, 3 * N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	calculateNewPositions << <100, 1000 >> >(dev_ballTable);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(ballTable, dev_ballTable, 3 * N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	Error:
		hipFree(dev_ballTable);

	return cudaStatus;
}

__device__ int detectCollision(GLfloat x, GLfloat y, GLfloat z, int ballNumber, GLfloat * ballTable){
	int collisionBall = -1;
	int num = ballNumber;
	for (int i = 0; i < N; i++){
		if (i != ballNumber){
			/*local*/
			GLfloat secondBallX = ballTable[i * 3];
			GLfloat secondBallY = ballTable[i * 3 + 1];
			GLfloat secondBallZ = ballTable[i * 3 + 2];
			GLfloat firstBallX = ballTable[ballNumber * 3];
			GLfloat firstBallY = ballTable[ballNumber * 3 + 1];
			GLfloat firstBallZ = ballTable[ballNumber * 3 + 2];
			GLfloat leftSide = (2 * RADIUS)*(2 * RADIUS);
			GLfloat rightSide = ((firstBallX - secondBallX)*(firstBallX - secondBallX) + (firstBallY - secondBallY)*(firstBallY - secondBallY)) + (firstBallZ - secondBallZ)*(firstBallZ - secondBallZ);
			/**/
			if (leftSide > rightSide)
			{
				//collisionBall = ballsMatrix[coordinateX + i][coordinateY + j];
				if (collisionMatrix[ballNumber + i*N] == false){
					collisionMatrix[ballNumber + i*N] = true;
					collisionMatrix[i + N * ballNumber] = true;
					collisionSafetyCounter[ballNumber + i*N] = 2;
					collisionSafetyCounter[i + N * ballNumber] = 2;
					return i;
				}
			}
			else{
				if (collisionSafetyCounter[ballNumber + i*N] > 0){
					collisionSafetyCounter[ballNumber + i*N] --;
					collisionSafetyCounter[i + N * ballNumber] --;
				}
				else{
					collisionMatrix[ballNumber + N * i] = false;
					collisionMatrix[i + N * ballNumber] = false;
				}
			}
		}
	}
	return -1;
}



/*********		CPU calculations		**************/
void calculateNewPositionsCPU(float* ballsTable){
	for(int k=0;k<N;k++){
			ballsTable[k * 3] += speedTable_H[k * 3];
			ballsTable[k * 3 + 1] += speedTable_H[k * 3 + 1];
			ballsTable[k * 3 + 2] += speedTable_H[k * 3 + 2];
			// Check if the ball exceeds the edges
			if (ballsTable[k * 3] > 1.0 - RADIUS){
				ballsTable[k * 3] = 1.0 - RADIUS;
				speedTable_H[k * 3] = -speedTable_H[k * 3] * SPRINGINESS;
			}
			if (ballsTable[k * 3] < -1.0 + RADIUS){
				ballsTable[k * 3] = -1.0 + RADIUS;
				speedTable_H[k * 3] = -speedTable_H[k * 3] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 1] > 1.0 - RADIUS){
				ballsTable[k * 3 + 1] = 1.0 - RADIUS;
				speedTable_H[k * 3 + 1] = -speedTable_H[k * 3 + 1] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 1] < -1.0 + RADIUS){
				ballsTable[k * 3 + 1] = -1.0 + RADIUS;
				speedTable_H[k * 3 + 1] = -speedTable_H[k * 3 + 1] * SPRINGINESS;
			}

			if (ballsTable[k * 3 + 2] > 1.0 - RADIUS){
				ballsTable[k * 3 + 2] = 1.0 - RADIUS;
				speedTable_H[k * 3 + 2] = -speedTable_H[k * 3 + 2] * SPRINGINESS;
			}
			if (ballsTable[k * 3 + 2] < -1.0 + RADIUS){
				ballsTable[k * 3 + 2] = -1.0 + RADIUS;
				speedTable_H[k * 3 + 2] = -speedTable_H[k * 3 + 2] * SPRINGINESS;
			}

			int ballDetected = detectCollisionCPU(ballsTable[k * 3], ballsTable[k * 3 + 1], ballsTable[k * 3 + 2], k, ballsTable);

			if (ballDetected != -1){
				float tmpSpeedX = speedTable_H[k * 3];
				float tmpSpeedY = speedTable_H[k * 3 + 1];
				float tmpSpeedZ = speedTable_H[k * 3 + 2];
				speedTable_H[k * 3] = speedTable_H[ballDetected * 3];
				speedTable_H[k * 3 + 1] = speedTable_H[ballDetected * 3 + 1];
				speedTable_H[k * 3 + 2] = speedTable_H[ballDetected * 3 + 2];
				speedTable_H[ballDetected * 3] = tmpSpeedX;
				speedTable_H[ballDetected * 3 + 1] = tmpSpeedY;
				speedTable_H[ballDetected * 3 + 2] = tmpSpeedZ;
			}
			//FRICTION
			if ((ballsTable[k * 3 + 1] < -1.0 + RADIUS + 0.0003) && (speedTable_H[k * 3 + 1] < 0.02)){
				speedTable_H[k * 3] *= 0.98;
				speedTable_H[k * 3 + 2] *= 0.98;
			}
			//gravity

			speedTable_H[k * 3 + 1] -= GRAVITY;
			//tmpSpeedTableY[k] -= 0.01f;
		}
}

int detectCollisionCPU(GLfloat x, GLfloat y, GLfloat z, int ballNumber, GLfloat * ballTable){
	int collisionBall = -1;
	int num = ballNumber;
	for (int i = 0; i < N; i++){
		if (i != ballNumber){
			/*local*/
			GLfloat secondBallX = ballTable[i * 3];
			GLfloat secondBallY = ballTable[i * 3 + 1];
			GLfloat secondBallZ = ballTable[i * 3 + 2];
			GLfloat firstBallX = ballTable[ballNumber * 3];
			GLfloat firstBallY = ballTable[ballNumber * 3 + 1];
			GLfloat firstBallZ = ballTable[ballNumber * 3 + 2];
			GLfloat leftSide = (2 * RADIUS)*(2 * RADIUS);
			GLfloat rightSide = ((firstBallX - secondBallX)*(firstBallX - secondBallX) + (firstBallY - secondBallY)*(firstBallY - secondBallY)) + (firstBallZ - secondBallZ)*(firstBallZ - secondBallZ);
			/**/
			if (leftSide > rightSide)
			{
				//collisionBall = ballsMatrix[coordinateX + i][coordinateY + j];
				if (collisionMatrix_H[ballNumber + i*N] == false){
					collisionMatrix_H[ballNumber + i*N] = true;
					collisionMatrix_H[i + N * ballNumber] = true;
					collisionSafetyCounter_H[ballNumber + i*N] = 2;
					collisionSafetyCounter_H[i + N * ballNumber] = 2;
					return i;
				}
			}
			else{
				if (collisionSafetyCounter_H[ballNumber + i*N] > 0){
					collisionSafetyCounter_H[ballNumber + i*N] --;
					collisionSafetyCounter_H[i + N * ballNumber] --;
				}
				else{
					collisionMatrix_H[ballNumber + N * i] = false;
					collisionMatrix_H[i + N * ballNumber] = false;
				}
			}
		}
	}
	return -1;
}

void specialKeys(int key, int x, int y) {

	//  Right arrow - increase rotation by 5 degree
	if (key == GLUT_KEY_RIGHT){
		yRotated += 1;
	}
		
	//  Left arrow - decrease rotation by 5 degree
	else if (key == GLUT_KEY_LEFT){
		yRotated -= 1;
	}
	else if (key == GLUT_KEY_UP){
		xRotated += 1;
	}

	else if (key == GLUT_KEY_DOWN){
		xRotated -= 1;
	}
	else if (key == GLUT_KEY_PAGE_UP){
		eyeZ += 0.05;
	}
	else if (key == GLUT_KEY_PAGE_DOWN){
		eyeZ -= 0.05;
	}

	//  Request display update
	//glutPostRedisplay();

}

